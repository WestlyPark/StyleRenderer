#include "hip/hip_runtime.h"
// Copyright (c) 2019, NVIDIA Corporation. All rights reserved.
//
// This work is made available under the Nvidia Source Code License-NC.
// To view a copy of this license, visit
// https://nvlabs.github.io/stylegan2/license.html

//#include <torch/types.h>
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
template <typename scalar_t>
static __global__ void fused_bias_act_kernel(
		scalar_t* out,
		const scalar_t* p_x,
		const scalar_t* p_b,
		const scalar_t* p_ref,
		int act, int grad, scalar_t alpha, scalar_t scale,
		int loop_x, int size_x, int step_b, int size_b, int use_bias, int use_ref) {
	int xi = blockIdx.x * loop_x * blockDim.x + threadIdx.x;
	scalar_t zero = 0.0;
	for(int loop_idx = 0; loop_idx < loop_x && xi < size_x; ++loop_idx, xi += blockDim.x) {
		scalar_t x = p_x[xi];
		if(use_bias)
			x += p_b[(xi / step_b) % size_b];
		scalar_t ref = use_ref ? p_ref[xi] : zero;
		scalar_t y;
		switch (act * 10 + grad) {
			default:
			case 10: y = x; break;
			case 11: y = x; break;
			case 12: y = 0.0; break;

			case 30: y = (x > 0.0) ? x : x * alpha; break;
			case 31: y = (ref > 0.0) ? x : x * alpha; break;
			case 32: y = 0.0; break;
		}
		out[xi] = y * scale;
	}
}
template <typename scalar_t>
static void fused_bias_act_cpu(
		scalar_t* out,
		const scalar_t* p_x,
		const scalar_t* p_b,
		const scalar_t* p_ref,
		int act, int grad, scalar_t alpha, scalar_t scale,
		int size_x, int step_b, int size_b, int use_bias, int use_ref) {
	scalar_t zero = 0.0;
	for(int xi = 0; xi < size_x; ++xi) {
		scalar_t x = p_x[xi];
		if(use_bias)
			x += p_b[(xi / step_b) % size_b];
		scalar_t ref = use_ref ? p_ref[xi] : zero;
		scalar_t y;
		switch (act * 10 + grad) {
			default:
			case 10: y = x; break;
			case 11: y = x; break;
			case 12: y = 0.0; break;

			case 30: y = (x > 0.0) ? x : x * alpha; break;
			case 31: y = (ref > 0.0) ? x : x * alpha; break;
			case 32: y = 0.0; break;
		}
		out[xi] = y * scale;
	}
}
bool fused_bias_act_op(float *y,
		const float *x,
		const float *b,
		const float *ref,
		int act, int grad, float alpha, float scale,
		int size_x, int step_b, int size_b, int use_bias, int use_ref, int use_cuda) {
	if(use_cuda) {
		int curDevice = -1;
		hipGetDevice(&curDevice);
		hipStream_t stream = c10::cuda::getCurrentCUDAStream(curDevice);
		int loop_x = 4;
		int block_size = 4 * 32;
		int grid_size = (size_x - 1) / (loop_x * block_size) + 1;
//		AT_DISPATCH_FLOATING_TYPES_AND_HALF(
//		torch::ScalarType::Float, "fused_bias_act_kernel", [&] {
		fused_bias_act_kernel<float><<<grid_size, block_size, 0, stream>>>(
			y, x, b, ref,
			act,
			grad,
			alpha,
			scale,
			loop_x,
			size_x,
			step_b,
			size_b,
			use_bias,
			use_ref);
//		});
	} else	fused_bias_act_cpu<float>(
			y, x, b, ref,
			act,
			grad,
			alpha,
			scale,
			size_x,
			step_b,
			size_b,
			use_bias,
			use_ref);

	return true;
}
